#include "hip/hip_runtime.h"
#include "cudarray/common.hpp"
#include "cudarray/nnet/one_hot.hpp"

namespace cudarray {

template <typename T>
__global__ void kernel_one_hot_encode(const int *labels, int n_classes, int n,
                                      T *out) {
  CUDA_GRID_STRIDE_LOOP(idx, n*n_classes) {
    int class_idx = idx % n_classes;
    int label_idx = idx / n_classes;
    out[idx] = labels[label_idx] == class_idx ? 1.0 : 0.0;
  }
}

template <typename T>
void one_hot_encode(const int *labels, int n_classes, int n, T *out) {
  kernel_one_hot_encode<<<cuda_blocks(n*n_classes), kNumBlockThreads>>>(
      labels, n_classes, n, out);
  CUDA_KERNEL_CHECK;
}

template void one_hot_encode(const int *labels, int n_classes, int n,
                             float *out);

// copy rows

template <typename T>
__global__ void kernel_copy_rows(const int *rowids, int numrows, int rowsize,
                                      T *from_mat, T *to_mat) {
  CUDA_GRID_STRIDE_LOOP(idx, numrows*rowsize) {
	int row = idx / rowsize;
	int column = idx % rowsize;
	int from_row = rowids[row];
    to_mat[idx] = from_mat[from_row * rowsize + column];
  }
}

template <typename T>
__global__ void kernel_copy_rows_mapto(const int *rowids, int numrows, int rowsize,
                                      T *from_mat, T *to_mat) {
  CUDA_GRID_STRIDE_LOOP(idx, numrows*rowsize) {
	int row = idx / rowsize;
	int column = idx % rowsize;
	int from_row = rowids[row];
    to_mat[from_row * rowsize + column] = from_mat[idx];
  }
}

template <typename T>
void copy_rows(const int *rowids, int numrows, int rowsize, T *from_mat, T *to_mat, 
		int mapfrom) {
  if (mapfrom == 1) {
	kernel_copy_rows<<<cuda_blocks(numrows*rowsize), kNumBlockThreads>>>(
		rowids, numrows, rowsize, from_mat, to_mat);
	CUDA_KERNEL_CHECK;
  }
  else {
	kernel_copy_rows_mapto<<<cuda_blocks(numrows*rowsize), kNumBlockThreads>>>(
		rowids, numrows, rowsize, from_mat, to_mat);
	CUDA_KERNEL_CHECK;
  }
}

template void copy_rows(const int *rowids, int numrows, int rowsize, float *from_mat,
                             float *to_mat, int mapfrom);

// copy sum rows


template <typename T>
__global__ void kernel_copy_sum_rows(const int *rowids, T *coefficients, int
		numsums, int numrows, int rowsize, T *from_mat, T *to_mat, float constant, 
		float var) {
  CUDA_GRID_STRIDE_LOOP(idx, numrows*rowsize) {
	int row = idx / rowsize;
	int column = idx % rowsize;
	to_mat[idx]=0; 
	int from_row=0;
	if (coefficients != NULL ){
		for (int j=0; j < numsums; j++) {
			from_row = rowids[row * numsums + j];
			to_mat[idx] += coefficients[row * numsums + j] * 
							from_mat[from_row * rowsize + column];
		}
	}
	else {
		for (int j=0; j < numsums; j++) {
			from_row = rowids[row * numsums + j];
			to_mat[idx] +=  constant * pow(var, (float)j) * from_mat[from_row *
				rowsize + column];
		}
	}
  }
}

template <typename T>
__global__ void kernel_copy_sum_rows_mapto(const int *rowids, T *coefficients, int
		numsums, int numrows, int rowsize, T *from_mat, T *to_mat, float constant, 
		float var) {
  CUDA_GRID_STRIDE_LOOP(idx, numrows*rowsize) {
	int row = idx / rowsize;
	int column = idx % rowsize;
	//to_mat[idx]=0; 
	int from_row=0;
	if (coefficients != NULL ){
		for (int j=0; j < numsums; j++) {
			from_row = rowids[row * numsums + j];
			// we have to use an atomic add here because different threads are
			// writnig to the same memory location
			atomicAdd(&to_mat[from_row * rowsize + column], 
					coefficients[row * numsums + j] * from_mat[idx]);
		}
	}
	else {
		for (int j=0; j < numsums; j++) {
			from_row = rowids[row * numsums + j];
			// we have to use an atomic add here because different threads are
			// writnig to the same memory location
			atomicAdd(&to_mat[from_row * rowsize + column], constant *
					pow(var,(float)j)  *from_mat[idx]);
		}
	}
  }
}

template <typename T>
void copy_sum_rows(const int *rowids, int numsums, int numrows, int rowsize, 
		T *from_mat, T *to_mat, int mapfrom, T *coefficients, float constant, 
		float var) {
  if (mapfrom == 1) {
	kernel_copy_sum_rows<<<cuda_blocks(numrows*rowsize), kNumBlockThreads>>>(
		rowids, coefficients, numsums, numrows, rowsize, from_mat, to_mat,
		constant, var);
	CUDA_KERNEL_CHECK;
  }
  else {
	kernel_copy_sum_rows_mapto<<<cuda_blocks(numrows*rowsize), kNumBlockThreads>>>(
		rowids, coefficients, numsums, numrows, rowsize, from_mat, to_mat,
		constant, var);
	CUDA_KERNEL_CHECK;
  }
}

template void copy_sum_rows(const int *rowids, int numsums, int numrows, int
		rowsize, float *from_mat, float *to_mat, int mapfrom, 
		float *coefficients, float constant, float var);

}
